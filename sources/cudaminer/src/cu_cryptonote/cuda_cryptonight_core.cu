#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cudadefs.h"

#ifndef _WIN32
#include <sys/time.h>
#include <unistd.h>
#else
#include <Windows.h>
#endif

extern int device_arch[8][2];
extern int device_bfactor[8];
extern int device_bsleep[8];

#include "cuda_cryptonight_aes.cuh"

__device__ __forceinline__ uint64_t cuda_mul128(uint64_t multiplier, uint64_t multiplicand, uint64_t* product_hi)
{
    *product_hi = __umul64hi(multiplier, multiplicand);
    return(multiplier * multiplicand);
}

__global__ void cryptonight_core_gpu_phase1(int threads, uint32_t *long_state, struct cryptonight_gpu_ctx *ctx)
{
  __shared__ uint32_t sharedMemory[1024];

    cn_aes_gpu_init(sharedMemory);

  __syncthreads();

    int thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 3;
    int sub = (threadIdx.x & 7) << 2;

    if (thread < threads)
    {
        uint32_t key[40], text[4], i;

        MEMCPY8(key, ctx[thread].key1, 20);
        MEMCPY8(text, &ctx[thread].state[sub + 16], 2);

        for(i = 0; i < 0x80000; i += 32)
        {
            cn_aes_pseudo_round_mut(sharedMemory, text, key);
            MEMCPY8(&long_state[(thread << 19) + sub + i], text, 2);
        }
    }
}

__global__ void cryptonight_core_gpu_phase2(int threads, int bfactor, int partidx, uint32_t *d_long_state, struct cryptonight_gpu_ctx *d_ctx)
{
  __shared__ uint32_t sharedMemory[1024];

    cn_aes_gpu_init(sharedMemory);

  __syncthreads();

#if __CUDA_ARCH__ >= 300

    int thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 2;
    int sub = threadIdx.x & 3;

    if (thread < threads)
    {
        int i, j, k;
        int batchsize = ITER >> (2+bfactor);
        int start = partidx * batchsize;
        int end = start + batchsize;
        uint32_t * __restrict__ long_state = &d_long_state[thread << 19];
        struct cryptonight_gpu_ctx *ctx = &d_ctx[thread];
        uint32_t a, b, c, x[4];
        uint32_t t1[4], t2[4], res;
        uint64_t reshi, reslo;

        a = ctx->a[sub];
        b = ctx->b[sub];

        for (i = start; i < end; ++i) {

            //j = ((uint32_t *)a)[0] & 0x1FFFF0;
            j = (__shfl((int)a, 0, 4) & 0x1FFFF0) >> 2;

            //cn_aes_single_round(sharedMemory, &long_state[j], c, a);
            x[0] = long_state[j + sub];
            x[1] = __shfl((int)x[0], sub+1, 4);
            x[2] = __shfl((int)x[0], sub+2, 4);
            x[3] = __shfl((int)x[0], sub+3, 4);
            c = a ^
                t_fn0(x[0] & 0xff) ^
                t_fn1((x[1] >> 8) & 0xff) ^
                t_fn2((x[2] >> 16) & 0xff) ^
                t_fn3((x[3] >> 24) & 0xff);

            //XOR_BLOCKS_DST(c, b, &long_state[j]);
            long_state[j + sub] = c ^ b;

            //MUL_SUM_XOR_DST(c, a, &long_state[((uint32_t *)c)[0] & 0x1FFFF0]);
            j = (__shfl((int)c, 0, 4) & 0x1FFFF0) >> 2;
            for( k = 0; k < 2; k++ ) t1[k] = __shfl((int)c, k, 4);
            for( k = 0; k < 4; k++ ) t2[k] = __shfl((int)a, k, 4);
            asm(
                "mad.lo.u64 %0, %2, %3, %4;\n\t"
                "mad.hi.u64 %1, %2, %3, %5;\n\t"
                : "=l"(reslo), "=l"(reshi)
                : "l"(((uint64_t *)t1)[0]), "l"(((uint64_t *)long_state)[j >> 1]), "l"(((uint64_t *)t2)[1]), "l"(((uint64_t *)t2)[0]));
            res = (sub & 2 ? reslo : reshi) >> (sub&1 ? 32 : 0);
            a = long_state[j + sub] ^ res;
            long_state[j + sub] = res;

            //j = ((uint32_t *)a)[0] & 0x1FFFF0;
            j = (__shfl((int)a, 0, 4) & 0x1FFFF0) >> 2;

            //cn_aes_single_round(sharedMemory, &long_state[j], b, a);
            x[0] = long_state[j + sub];
            x[1] = __shfl((int)x[0], sub+1, 4);
            x[2] = __shfl((int)x[0], sub+2, 4);
            x[3] = __shfl((int)x[0], sub+3, 4);
            b = a ^
                t_fn0(x[0] & 0xff) ^
                t_fn1((x[1] >> 8) & 0xff) ^
                t_fn2((x[2] >> 16) & 0xff) ^
                t_fn3((x[3] >> 24) & 0xff);

            //XOR_BLOCKS_DST(b, c, &long_state[j]);
            long_state[j + sub] = c ^ b;

            //MUL_SUM_XOR_DST(b, a, &long_state[((uint32_t *)b)[0] & 0x1FFFF0]);
            j = (__shfl((int)b, 0, 4) & 0x1FFFF0) >> 2;
            for( k = 0; k < 2; k++ ) t1[k] = __shfl((int)b, k, 4);
            for( k = 0; k < 4; k++ ) t2[k] = __shfl((int)a, k, 4);
            asm(
                "mad.lo.u64 %0, %2, %3, %4;\n\t"
                "mad.hi.u64 %1, %2, %3, %5;\n\t"
                : "=l"(reslo), "=l"(reshi)
                : "l"(((uint64_t *)t1)[0]), "l"(((uint64_t *)long_state)[j >> 1]), "l"(((uint64_t *)t2)[1]), "l"(((uint64_t *)t2)[0]));
            res = (sub & 2 ? reslo : reshi) >> (sub&1 ? 32 : 0);
            a = long_state[j + sub] ^ res;
            long_state[j + sub] = res;
        }

        if( bfactor > 0 ) {

            ctx->a[sub] = a;
            ctx->b[sub] = b;
        }
    }

#else // __CUDA_ARCH__ < 300

    int thread = blockDim.x * blockIdx.x + threadIdx.x;

    if (thread < threads)
    {
        int i, j;
        int batchsize = ITER >> (2+bfactor);
        int start = partidx * batchsize;
        int end = start + batchsize;
        uint32_t *long_state = &d_long_state[thread << 19];
        struct cryptonight_gpu_ctx *ctx = &d_ctx[thread];
        uint32_t a[4], b[4], c[4];

        MEMCPY8(a, ctx->a, 2);
        MEMCPY8(b, ctx->b, 2);

        for (i = start; i < end; ++i) {

            j = (a[0] & 0x1FFFF0) >> 2;
            cn_aes_single_round(sharedMemory, (uint8_t *)&long_state[j], c, a);
            XOR_BLOCKS_DST(c, b, &long_state[j]);
            MUL_SUM_XOR_DST(c, a, (uint8_t *)&long_state[(c[0] & 0x1FFFF0) >> 2]);
            j = (a[0] & 0x1FFFF0) >> 2;
            cn_aes_single_round(sharedMemory, (uint8_t *)&long_state[j], b, a);
            XOR_BLOCKS_DST(b, c, &long_state[j]);
            MUL_SUM_XOR_DST(b, a, &long_state[(b[0] & 0x1FFFF0) >> 2]);
        }

        if( bfactor > 0 ) {

            MEMCPY8(ctx->a, a, 2);
            MEMCPY8(ctx->b, b, 2);
        }
    }

#endif // __CUDA_ARCH__ >= 300
}

__global__ void cryptonight_core_gpu_phase3(int threads, uint32_t *long_state, struct cryptonight_gpu_ctx *ctx)
{
  __shared__ uint32_t sharedMemory[1024];

    cn_aes_gpu_init(sharedMemory);

  __syncthreads();

    int thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 3;
    int sub = (threadIdx.x & 7) << 2;

    if (thread < threads)
    {
        uint32_t key[40], text[4], i, j;
        MEMCPY8(key, ctx[thread].key2, 20);
        MEMCPY8(text, &ctx[thread].state[sub + 16], 2);

        for (i = 0; i < 0x80000; i += 32)
        {
            for(j = 0; j < 4; ++j)
                text[j] ^= long_state[(thread << 19) + sub + i + j];

            cn_aes_pseudo_round_mut(sharedMemory, text, key);
        }

        MEMCPY8(&ctx[thread].state[sub + 16], text, 2);
    }
}


__host__ bool cryptonight_core_cpu_init(int deviceNo)
{
  hipSetDevice(deviceNo);
  cn_aes_cpu_init();
    return true;
}

__host__ void cryptonight_core_cpu_hash(int deviceNo, int blocks, int threads, uint32_t *d_long_state, struct cryptonight_gpu_ctx *d_ctx, int device_bfactor, int cuda_major)
{
    hipSetDevice(deviceNo);
    dim3 grid(blocks);
    dim3 block(threads);
    dim3 block4(threads << 2);
    dim3 block8(threads << 3);

    size_t shared_size = 1024;
    int i, partcount = 1 << device_bfactor;

    cryptonight_core_gpu_phase1<<<grid, block8, shared_size>>>(blocks*threads, d_long_state, d_ctx);
    hipDeviceSynchronize();
    if( partcount > 1 ) usleep(100);

    for( i = 0; i < partcount; i++ ) {
        cryptonight_core_gpu_phase2<<<grid, (cuda_major >= 3 ? block4 : block), shared_size>>>(blocks*threads, device_bfactor, i, d_long_state, d_ctx);
        hipDeviceSynchronize();
        if( partcount > 1 ) usleep(100);
    }

    cryptonight_core_gpu_phase3<<<grid, block8, shared_size>>>(blocks*threads, d_long_state, d_ctx);
    hipDeviceSynchronize();
}

#ifdef _MSC_VER
void usleep(__int64 waitTime)
{
    if (waitTime > 0)
    {
        if (waitTime > 100)
        {
            // use a waitable timer for larger intervals > 0.1ms

            HANDLE timer;
            LARGE_INTEGER ft;

            ft.QuadPart = -(10*waitTime); // Convert to 100 nanosecond interval, negative value indicates relative time

            timer = CreateWaitableTimer(NULL, TRUE, NULL);
            SetWaitableTimer(timer, &ft, 0, NULL, NULL, 0);
            WaitForSingleObject(timer, INFINITE);
            CloseHandle(timer);
        }
        else
        {
            // use a polling loop for short intervals <= 100ms

            LARGE_INTEGER perfCnt, start, now;
            __int64 elapsed;

            QueryPerformanceFrequency(&perfCnt);
            QueryPerformanceCounter(&start);
            do {
                QueryPerformanceCounter((LARGE_INTEGER*) &now);
                elapsed = (__int64)((now.QuadPart - start.QuadPart) / (float)perfCnt.QuadPart * 1000 * 1000);
            } while ( elapsed < waitTime );
        }
    }
}
#endif
